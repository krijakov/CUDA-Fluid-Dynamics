#include "hip/hip_runtime.h"
#include "simulation.cuh"

__global__ void step_kernel(Particle* particles, Velocity* velocities, int count, float dt){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count){
        particles[idx].x += velocities[idx].vx * dt;
        particles[idx].y += velocities[idx].vy * dt;
        particles[idx].z += velocities[idx].vz * dt;
    }
}

void simulate_step_gpu(Particle* particles, Velocity* velocities, int count, float dt){
    Particle* d_particles;
    Velocity* d_velocities;
    size_t size = count * sizeof(Particle);
    size_t vel_size = count * sizeof(Velocity);

    hipMalloc(&d_particles, size);
    hipMalloc(&d_velocities, vel_size);

    hipMemcpy(d_particles, particles, size, hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, velocities, vel_size, hipMemcpyHostToDevice);

    step_kernel<<<(count + 255) / 256, 256>>>(d_particles, d_velocities, count, dt);
    hipDeviceSynchronize();

    hipMemcpy(particles, d_particles, size, hipMemcpyDeviceToHost);
    hipMemcpy(velocities, d_velocities, vel_size, hipMemcpyDeviceToHost);

    hipFree(d_particles);
    hipFree(d_velocities);
}